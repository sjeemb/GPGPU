#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "SyncedMemory.h"

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

const int W = 40;
const int H = 12;

__global__ void Draw(char *frame) {
	// TODO: draw more complex things here
	// Do not just submit the original file provided by the TA!
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (y < H and x < W) {
		char c;
		if (x == W-1) {
			c = y == H-1 ? '\0' : '\n';
		} else if (y == 0 or y == H-1 or x == 0 or x == W-2) {
			c = ':';
                }  else if ((y == 5 && (x >= 18 && x <= 21))||
                            (y == 6 && (x >= 16 && x <= 21))||
                            (y == 7 && (x >= 14 && x <= 21))||
                            (y == 8 && (x >= 12 && x <= 21))||
                            (y == 9 && (x >= 10 && x <= 21))||
                            (y == 10 && (x >= 8 && x <= 21))) {
                        c = '#';
                }  else if (x == 32 && y == 5) {
                        c = '<';
                }  else if (x == 33 && (y >= 5 && y <= 9)) {
                        c = '|';
                }  else if (x == 33 && y == 10) {
                        c = '#';
		} else {
			c = ' ';
		}
		frame[y*W+x] = c;
	}
}

int main(int argc, char **argv)
{
	MemoryBuffer<char> frame(W*H);
	auto frame_smem = frame.CreateSync(W*H);
	CHECK;

	Draw<<<dim3((W-1)/16+1,(H-1)/12+1), dim3(16,12)>>>(frame_smem.get_gpu_wo());
	CHECK;

	puts(frame_smem.get_cpu_ro());
	CHECK;
	return 0;
}
